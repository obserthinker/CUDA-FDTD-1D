#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "compute.cuh"
#include "Hy.cuh"
#include "Ex.cuh"
#include "boundary.cuh"
#include "source.cuh"
#include "save2file.cuh"

using namespace std;

extern float *Ex, *Hy;
extern int step_time;

void compute()
{
	int i;
	for (i = 0; i < step_time; i++){
		Hy_cmp();
		Ex_cmp();
		boundary_cmp_MUR();
		//Boundary_cmp_PEC();
		src_cmp(i);
		//save2file();
	}
}