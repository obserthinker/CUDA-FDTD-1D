﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>
#include <fstream>
#include "input.cuh"
#include "compute.cuh"

using namespace std;

void Boundary_PEC(float* Ex, int Nx);
void Save2File(float* Ex, float* Hy, int Nx);

float *Ex, *Hy;
int step_space, step_time;
float coe_Ex, coe_Hy, coe_MUR, dt, dz, Ex_nbd, Ex_bd;

const float PI = 3.141592653589793;
const float C = 299792458;
const float mu = (4 * PI)*1e-7;
const float epsilon = 8.85e-12;

void main()
{
	input();
	compute();
}