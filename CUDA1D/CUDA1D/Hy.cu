#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "Hy.cuh"

extern float *Ex, *Hy;
extern int step_space, step_time;
extern float coe_Hy, dt, dz;
extern const float mu;

void Hy_init()
{
	int i, step_space_Hy;
	step_space_Hy = step_space;

	Hy = (float *)malloc(step_space_Hy * sizeof(float));

	for (i = 0; i < step_space_Hy; i++)	{
		Hy[i] = 0.f;
	}

	coe_Hy = dt / (mu*dz);
}

void Hy_cmp()
{
	int i;
	for (i = 0; i < step_space; i++){
		Hy[i] = Hy[i] - coe_Hy*(Ex[i + 1] - Ex[i]);
	}
}