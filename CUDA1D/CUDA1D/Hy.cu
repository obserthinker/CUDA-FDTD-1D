#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "Hy.cuh"

extern float *Ex, *Hy;
extern int step_space, step_time;
extern float *coe_Hy, dt, dz;
extern const float mu;
//
extern float *dev_Ex, *dev_Hy, *dev_coe_Hy;

void Hy_transfer_Host_Device(int size)
{
	hipMemcpy(dev_Hy, Hy, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_coe_Hy, coe_Hy, sizeof(float), hipMemcpyHostToDevice);
}

void Hy_init()
{
	int i, step_space_Hy;
	step_space_Hy = step_space;

	//Allocate memory for Host
	Hy = (float *)malloc(step_space_Hy * sizeof(float));
	coe_Hy = (float *)malloc(sizeof(float));
	//Allocate memory for device
	hipMalloc(&dev_coe_Hy, step_space_Hy * sizeof(float));
	hipMalloc(&dev_coe_Hy, sizeof(float));

	for (i = 0; i < step_space_Hy; i++)	{
		Hy[i] = 0.f;
	}

	*coe_Hy = dt / (mu*dz);

	Hy_transfer_Host_Device(step_space_Hy);
}

void Hy_cmp()
{
	int i;
	for (i = 0; i < step_space; i++){
		Hy[i] = Hy[i] - (*coe_Hy)*(Ex[i + 1] - Ex[i]);
	}
}