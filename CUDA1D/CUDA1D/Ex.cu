#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "Ex.cuh"

using namespace std;

extern float *Ex, *Hy;
extern int step_space, step_time;
extern float *coe_Ex, dt,dz;
const float epsilon = 8.85e-12;
//
extern float *dev_Ex, *dev_Hy, *dev_coe_Ex;

void Ex_init()
{
	int i;
	int step_space_Ex;

	step_space_Ex = step_space + 1;
	//allocate memory for host
	Ex = (float *)malloc(step_space_Ex * sizeof(float));
	coe_Ex = (float *)malloc(sizeof(float));
	//allocate memory for device
	hipMalloc(&dev_Ex, step_space_Ex * sizeof(float));
	hipMalloc(&dev_coe_Ex, sizeof(float));

	for (i = 0; i < step_space_Ex; i++){
		Ex[i] = 0.f;
	}

	*coe_Ex = dt / (epsilon * dz);
}

void Ex_cmp()
{
	int i;
	for (i = 1; i < step_space; i++){
		Ex[i] = Ex[i] - (*coe_Ex)*(Hy[i] - Hy[i - 1]);
	}
}