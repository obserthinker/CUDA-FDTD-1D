#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "boundary.cuh"

extern float Ex_bd, dz, dt, coe_MUR, Ex_nbd;
extern float *Ex;
const float C = 3e8;
extern int step_space;

void boundary_init()
{
	Ex_nbd = 0.f;
	Ex_bd = 0.f;

	coe_MUR = (C*dt - dz) / (C*dt + dz);
}

void boundary_cmp_MUR()
{
	int bd, nbd;
	bd = step_space;
	nbd = bd - 1;

	Ex[bd] = Ex_nbd + coe_MUR*(Ex[nbd] - Ex_bd);

	Ex_bd = Ex[bd];
	Ex_nbd = Ex[nbd];
}

void Boundary_cmp_PEC()
{
	Ex[step_space] = 0.f;
}