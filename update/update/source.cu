#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include "source.cuh"

extern float dz, dt;
extern float *Ex;
const float C = 3e8;

void	src_init()
{
	dz = 0.015;
	dt = dz / (2 * C);
}

void src_cmp(int current_timestep)
{
	float T, T0, vt, val_src, time;

	time = current_timestep * dt;
	T = 5e-10;
	T0 = 3 * T;
	vt = (time - T0) / T;

	val_src = exp(-pow(vt, 2));

	Ex[0] = val_src;
}