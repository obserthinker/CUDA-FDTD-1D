#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "Hy.cuh"
#include "Ex.cuh"
#include "src.cuh"
#include "save2file.cuh"

using namespace std;

//host
float *Ex, *Hy, coe_Hy, coe_Ex, dt, dz;
int size_space, size_time, size_Hy, size_Ex;
//device
float *dev_Ex, *dev_Hy;
//test
int gpu_data_int;

int main()
{
	size_space = 30;
	size_time = 300;
	/***** source *****/
	src_init();

	/*********  Hy ********/
	Hy_init(size_space);
	//Hy_checkout(size_Hy);
	Hy_transfer_host_device(size_Hy);

	/******** Ex ********/
	Ex_init(size_space);
	//Ex_checkout(size_Ex);
	Ex_transfer_host_device(size_Ex);

	/******** File ********/
	file_init();

	for (int i = 0; i < size_time; i++)
	{
		Hy_cmp_kernel << <1, 1 >> >(dev_Hy, dev_Ex, coe_Hy, size_space);
		Ex_cmp_kernel << <1, 1 >> >(dev_Hy, dev_Ex, coe_Ex, size_space);
		Ex_boundary_PEC_kernel << <1, 1 >> >(dev_Ex, size_space);
		src_cmp_kernel << <1, 1 >> >(i, dt, dev_Ex, size_space);

		Hy_transfer_device_host(size_Hy);
		Ex_transfer_device_host(size_Ex);
		
		save2file();
	}

	//Hy_checkout(size_Hy);
	//Ex_checkout(size_Ex);
	
    return 0;
}