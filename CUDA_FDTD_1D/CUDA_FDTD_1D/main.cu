#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "Hy.cuh"
#include "Ex.cuh"
#include "src.cuh"
#include "save2file.cuh"

#define MULTI_B_T

using namespace std;

//host
float *Ex, *Hy, coe_Hy, coe_Ex, dt, dz, coe_MUR, Ex_bd, Ex_nbd;
int size_space, size_time, size_Hy, size_Ex;
//device
float *dev_Ex, *dev_Hy, *dev_Ex_bd, *dev_Ex_nbd;
//test
int gpu_data_int;

int main()
{
	size_space = 30;
	size_time = 300;
	/***** source *****/
	src_init();

	/*********  Hy ********/
	Hy_init(size_space);
	//Hy_checkout(size_Hy);
	Hy_transfer_host_device(size_Hy);

	/******** Ex ********/
	Ex_init(size_space);
	//Ex_checkout(size_Ex);
	Ex_transfer_host_device(size_Ex);

	/******** File ********/
	file_init();

	for (int i = 0; i < size_time; i++)
	{
#ifndef MULTI_B_T
		Hy_cmp_kernel << <1, 1 >> >(dev_Hy, dev_Ex, coe_Hy, size_space);
		Ex_cmp_kernel << <1, 1 >> >(dev_Hy, dev_Ex, coe_Ex, size_space);
		Ex_boundary_PEC_kernel << <1, 1 >> >(dev_Ex, size_space);
		src_cmp_kernel << <1, 1 >> >(i, dt, dev_Ex, size_space);
#else
		Hy_cmp_kernel << <4, 10 >> >(dev_Hy, dev_Ex, coe_Hy, size_space);
		Ex_cmp_kernel << <4, 10 >> >(dev_Hy, dev_Ex, coe_Ex, size_space);
		//PEC
		//Ex_boundary_PEC_kernel << <1, 1 >> >(dev_Ex, size_space);
		//MUR
		Ex_boundary_MUR_kernek << <1, 1 >> >(dev_Ex, size_space, dev_Ex_bd, dev_Ex_nbd, coe_MUR);
		src_cmp_kernel << <1, 1 >> >(i, dt, dev_Ex, size_space);
#endif
		Hy_transfer_device_host(size_Hy);
		Ex_transfer_device_host(size_Ex);
		
		save2file();
	}

	hipFree(dev_Ex);
	hipFree(dev_Hy);
	//Hy_checkout(size_Hy);
	//Ex_checkout(size_Ex);
	
    return 0;
}