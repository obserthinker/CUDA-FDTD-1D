#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>

using namespace std;

extern int size_space;
extern float *Ex, *Hy;

void file_init()
{
	fstream outEx, outHy;

	outEx.open("Ex.txt", ios::out);
	outEx.close();
	outHy.open("Hy.txt", ios::out);
	outHy.close();
}

void save2file()
{
	fstream outEx;
	outEx.open("Ex.txt", ios::app);

	int i;

	for  ( i = 0; i < size_space + 1; i++){
		outEx << Ex[i] << "\t";
	}
	outEx << endl << endl;
	outEx.close();

	fstream outHy;
	outHy.open("Hy.txt", ios::app);

	for ( i = 0; i < size_space; i++){
		outHy << Hy[i] << "\t";
	}
	outHy << endl << endl;
	outHy.close();
}