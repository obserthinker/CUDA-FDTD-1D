#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "Hy.cuh"
#include "Ex.cuh"

using namespace std;

extern int step_space, size_Ex, size_Hy;
extern float *dev_Hy, *dev_Ex;

void cmp_compute();
void cmp_show_result();

void cmp()
{
	cmp_compute();

	cmp_show_result();
}

void cmp_compute()
{
	Hy_cmp_kernel << <1, 1 >> >(dev_Hy, dev_Ex, step_space);
	//Ex_cmp_kernel << <1, 1 >> >(dev_Hy, dev_Ex, step_space);
}

void cmp_show_result()
{
	Hy_transfer_device_host(size_Hy);
	Hy_checkout(size_Ex);
	//Ex_transfer_device_host(size_Ex);
	//Ex_checkout(size_Ex);
}