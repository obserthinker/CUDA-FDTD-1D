#include "hip/hip_runtime.h"

#include <iostream>

using namespace std;

//host
extern float *Hy, coe_Hy, dt, dz;
extern int size_space, size_Hy;
const float PI = 3.141592653589793f;
const float mu = (4 * PI)*1e-7f;

//device
extern float *dev_Hy, *dev_Ex;

void Hy_init_malloc(int );
void Hy_init_assignValue(int );
void Hy_checkout();
void Hy_transfer_host_device();
void Hy_transfer_device_host();

void Hy_init(int space_size)
{
	size_Hy = space_size;
	Hy_init_malloc(size_Hy);
	Hy_init_assignValue(size_Hy);
}

void Hy_init_malloc(int size)
{
	//host
	Hy = (float *)malloc(size * sizeof(float));
	//device
	hipMalloc(&dev_Hy, size * sizeof(float));
}

void Hy_init_assignValue(int size)
{
	int i;
	for ( i = 0; i < size; i++){
		Hy[i] = 0.f;
	}
	
	coe_Hy = dt / (mu * dz);
}

void Hy_checkout(int size)
{
	cout << "Hy: size = " << size << endl;
	cout << "coe_Hy = " << coe_Hy;
	cout << "Hy: ";
	for (int i = 0; i < size; i++)
	{
		cout << Hy[i] << "\t";
	}
	cout << endl;
}

void Hy_transfer_host_device(int size_Hy)
{
	hipMemcpy(dev_Hy, Hy, size_Hy * sizeof(float), hipMemcpyHostToDevice);
}

void Hy_transfer_device_host(int size_Hy)
{
	hipMemcpy(Hy, dev_Hy, size_Hy * sizeof(float), hipMemcpyDeviceToHost);
}

__global__ void Hy_cmp_kernel(float* dev_Hy, float * dev_Ex, float coe_Hy, int size_space)
{
	int i;
	for (i = 0; i < size_space; i++){
		dev_Hy[i] = dev_Hy[i] - (coe_Hy)*(dev_Ex[i + 1] - dev_Ex[i]);
		//test
		//dev_Hy[i] = i*10.0;
	}
}