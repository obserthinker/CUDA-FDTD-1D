#include "hip/hip_runtime.h"

#include <iostream>

using namespace std;

extern float *Ex, *dev_Ex, coe_Ex, dt, dz;
extern int size_space, size_Ex;
const float epsilon = 8.85e-12;

void Ex_init_allocate(int size_Ex)
{
	Ex = (float *)malloc(size_Ex* sizeof(float));
	hipMalloc(&dev_Ex, size_Ex* sizeof(float));
}

void Ex_init_assignValue(int size_Ex)
{
	for (int i = 0; i < size_Ex; i++){
		Ex[i] = 0.f;
	}

	coe_Ex = dt / (epsilon * dz);
}

void Ex_transfer_host_device(int size_Ex)
{
	hipMemcpy(dev_Ex, Ex, size_Ex * sizeof(float), hipMemcpyHostToDevice);
}

void Ex_transfer_device_host(int size_Ex)
{
	hipMemcpy(Ex, dev_Ex, size_Ex * sizeof(float), hipMemcpyDeviceToHost);
}

__global__ void Ex_cmp_kernel(float* dev_Hy, float * dev_Ex, float coe_Ex, int step_space)
{
	for (int i = 1; i < step_space; i++){
		dev_Ex[i] = dev_Ex[i] - coe_Ex * (dev_Hy[i] - dev_Hy[i - 1]);
		//test
		//dev_Ex[i] = i / 10.0;
	}
}

void Ex_checkout(int size)
{
	cout << "Ex: size = " << size << endl;
	cout << "coe_Ex = " << coe_Ex;
	cout << "Ex: ";
	for (int i = 0; i < size; i++)
	{
		cout << Ex[i] << "\t";
	}
	cout << endl;
}

void Ex_init(int size_space)
{
	size_Ex = size_space + 1;
	Ex_init_allocate(size_Ex);
	Ex_init_assignValue(size_Ex);
}

__global__ void Ex_boundary_PEC_kernel(float* dev_Ex, int size_space)
{
	dev_Ex[size_space] = 0.f;
}