#include "hip/hip_runtime.h"

#include <iostream>

#define MULTI_B_T

using namespace std;

extern float *Ex, *dev_Ex, coe_Ex, dt, dz, coe_MUR, Ex_bd, Ex_nbd, *dev_Ex_bd, *dev_Ex_nbd;
extern int size_space, size_Ex;
const float epsilon = 8.85e-12;
const float C = 3e8;

void Ex_init_allocate(int size_Ex)
{
	Ex = (float *)malloc(size_Ex* sizeof(float));
	hipMalloc(&dev_Ex, size_Ex* sizeof(float));
	hipMalloc(&dev_Ex_bd, sizeof(float));
	hipMalloc(&dev_Ex_nbd, sizeof(float));
}

void Ex_init_assignValue(int size_Ex)
{
	for (int i = 0; i < size_Ex; i++){
		Ex[i] = 0.f;
	}

	coe_Ex = dt / (epsilon * dz);

	Ex_nbd = 0.f;
	Ex_bd = 0.f;

	coe_MUR = (C * dt - dz) / (C * dt + dz);
}

void Ex_transfer_host_device(int size_Ex)
{
	hipMemcpy(dev_Ex, Ex, size_Ex * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_Ex_bd, &Ex_bd, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_Ex_nbd, &Ex_nbd, sizeof(float), hipMemcpyHostToDevice);
}

void Ex_transfer_device_host(int size_Ex)
{
	hipMemcpy(Ex, dev_Ex, size_Ex * sizeof(float), hipMemcpyDeviceToHost);
}

#ifndef MULTI_B_T
__global__ void Ex_cmp_kernel(float* dev_Hy, float * dev_Ex, float coe_Ex, int step_space)
{
	for (int i = 1; i < step_space; i++){
		dev_Ex[i] = dev_Ex[i] - coe_Ex * (dev_Hy[i] - dev_Hy[i - 1]);
		//test
		//dev_Ex[i] = i / 10.0;
	}
}
#else
__global__ void Ex_cmp_kernel(float* dev_Hy, float * dev_Ex, float coe_Ex, int step_space)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	for (tid = 1; tid < step_space; tid++){
		dev_Ex[tid] = dev_Ex[tid] - coe_Ex * (dev_Hy[tid] - dev_Hy[tid - 1]);
		//test
		//dev_Ex[i] = i / 10.0;
	}
}
#endif

void Ex_checkout(int size)
{
	cout << "Ex: size = " << size << endl;
	cout << "coe_Ex = " << coe_Ex;
	cout << "Ex: ";
	for (int i = 0; i < size; i++)
	{
		cout << Ex[i] << "\t";
	}
	cout << endl;
}

void Ex_init(int size_space)
{
	size_Ex = size_space + 1;
	Ex_init_allocate(size_Ex);
	Ex_init_assignValue(size_Ex);
}

__global__ void Ex_boundary_PEC_kernel(float* dev_Ex, int size_space)
{
	dev_Ex[size_space] = 0.f;
}

__global__ void Ex_boundary_MUR_kernek(float *dev_Ex, int size_space, float *dev_Ex_bd, float *dev_Ex_nbd, float coe_MUR)
{
	int bd, nbd;
	bd = size_space;
	nbd = bd - 1;

	dev_Ex[bd] = (*dev_Ex_nbd) + coe_MUR * (dev_Ex[nbd] - (*dev_Ex_bd));

	*dev_Ex_bd = dev_Ex[bd];
	*dev_Ex_nbd = dev_Ex[nbd];
}