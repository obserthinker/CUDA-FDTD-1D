#include "hip/hip_runtime.h"

#include <iostream>
#include <cmath>

using namespace std;

extern float dz, dt;
extern int size_space, size_time;
const float C = 3e8f;

void src_init()
{
	dz = 0.015;
	dt = dz / (2 * C);
}

void src_checkout()
{
	cout << "dz: " << dz << endl;
	cout << "dt: " << dt << endl;
}

__global__ void src_cmp_kernel(int current_timestep, float dt, float* dev_Ex, int size_space)
{
	float T, T0, vt, val_src, time;

	time = current_timestep * dt;

	T = 5e-10f;
	T0 = 3 * T;
	vt = (time - T0) / T;

	val_src = expf(-powf(vt, 2.0f));

	dev_Ex[0] = val_src;
}